#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <cstdlib>
#include <sstream>
#include <string>


#include "Data.h"


__device__ __host__ double StateEq(double rho, double Cg, double rho0, double c0, double Gam_lq, double p0, int var_Eq_state)
{
	double tmp;
		// return  c0*c0*(rho - rho0);
	switch (var_Eq_state)
	{
	case 0: { tmp = rho0*c0*c0 / Gam_lq * (pow(rho / (rho0*(1 - Cg)), Gam_lq) - 1.0) + p0;
		if (tmp < p0) tmp = p0; }; break;
	case 1: { tmp = rho0*c0*c0 * (rho / (rho0*(1 - Cg)) - 1.0) + p0;
		if (tmp < p0) tmp = p0; }; break;
	}

	return tmp;

	}

__device__ __host__ double StateEq1(double p, double Cg, double rho0, double c0, double Gam_lq, double p0, int var_Eq_state)
	{
		//return  rho0 + p / (c0*c0);
		switch (var_Eq_state)
		{
		case 0: {return rho0*(1 - Cg)*pow((p - p0)*Gam_lq / (rho0*c0*c0) + 1.0, 1.0 / Gam_lq); }; break;
		case 1: {return rho0*(1 - Cg)*( (p - p0)/ (rho0*c0*c0) + 1.0); }; break;
		}
	}

double FreeSurf(double T, double p)
{
//	double p_tmp;
//	if (T > T_surf) {return p0;}
//		else {return T* (p0-p) / (T_surf) + p;};
//	p_tmp = p / 1.2;
//	if (p_tmp < p0) {p_tmp = p0;};
	return p0;
}

int main()
{
	var_wall = 1;
	var_Eq_state = 1;
	out_num = 0;
	Tm = 0.0;
	T_all = 1.5;
	tau = 1.0e-6;
	T_out = 1000.0*tau;
	T_surf = 10.0*tau;
	hx = 0.25;
	hz = 1.0;
	Xm = 10.0;
	Zm = 1000.0;
	Z_max = 1.5*Zm;
	Z_ch = 7.5e3;

	Im = int(Xm / hx) + 1;
	J_max = int(Z_max / hz) + 1; 
	Jm = new int [Im];
	for (i = 0; i < Im; i++){
			Jm[i] = int(Zm/hz);
		}

	x = new double [Im];
	for (i = 0; i < Im; i++){
		x[i] = i*hx;
	}
	z = new double [J_max];
	for (j = 0; j < J_max; j++){
		z[j] = j*hz;
	}

	Ig = 5*Im;
	GrX = new double [Ig];
	GrZ = new double [Ig];
	for (i = 0; i < Ig; i++){
			GrX[i] = i * double (Xm / (Ig-1));
			GrZ[i] = Zm;
		}


	p = new double *[Im];
	Pg = new double *[Im];
	rho = new double *[Im];
	rho_n = new double *[Im];
	u = new double *[Im];
	v = new double *[Im];
	u_n = new double *[Im];
	v_n = new double *[Im];
	mu = new double *[Im];
	Cg = new double *[Im];
	Cp = new double *[Im];
	Cp0 = new double *[Im];
	Nb = new double *[Im];
	Rb = new double *[Im];
	dRb = new double *[Im];
	ddRb = new double *[Im];
	Eps = new double *[Im];
	Mg = new double *[Im];
	D_eff = new double *[Im];
	Nt = new double *[Im];
	Ind = new int *[Im];
	for (i = 0; i < Im; i++){
		p[i] = new double [J_max];
		Pg[i] = new double[J_max];
		rho[i] = new double [J_max];
		rho_n[i] = new double [J_max];
		u[i] = new double [J_max];
		v[i] = new double [J_max];
		u_n[i] = new double [J_max];
		v_n[i] = new double [J_max];
		mu[i] = new double [J_max];
		Cg[i] = new double [J_max];
		Cp[i] = new double [J_max];
		Cp0[i] = new double [J_max];
		Nb[i] = new double [J_max];
		Rb[i] = new double [J_max];
		dRb[i] = new double[J_max];
		ddRb[i] = new double[J_max];
		Eps[i] = new double [J_max];
		Mg[i] = new double [J_max];
		D_eff[i] = new double [J_max];
		Nt[i] = new double [J_max];
		Ind[i] = new int [J_max];
	}

	for (i = 0; i < Im; i++){
		for (j = 0; j < J_max; j++){
			p[i][j] = rho0 * g * (Z_ch - z[j]);	
			Pg[i][j] = p[i][j];
			Cp0[i][j] = Kh*sqrt(p[i][j]);
			Nb[i][j] = 0.0;// 1.0e13;
			Rb[i][j] = 0.0; // Rb_min;
			dRb[i][j] = 0.0;
			ddRb[i][j] = 0.0;
			Mg[i][j] = 0.0; // 4.0*Pi * Pg[i][j] *Rb[i][j]*Rb[i][j]*Rb[i][j]*MH2O / (3.0* Rg * T);
			Cp[i][j] = Cp0[i][j] - Nb[i][j] * Mg[i][j] / rho0;
			if (Cp[i][j] < 0) {Cp[i][j] = 1.0e-3;};
        	Cg[i][j] = Nb[i][j]* 4.0*Pi/3.0 *Rb[i][j]*Rb[i][j]*Rb[i][j];
        	//Cg[i][j] = Cg[i][j] / (1.0 + Cg[i][j]);
			rho[i][j] = StateEq1(p[i][j], Cg[i][j], rho0, c0, Gam_lq, p0, var_Eq_state);
       		E = E0*(1-12*Cp[i][j]);
       		mu[i][j] =mu0*exp(E/(kB*T));
		//	mu[i][j] = 1.0e7;
			u[i][j] = 0.0;
			v[i][j] = 0.0;
			u_n[i][j] = 0.0;
			v_n[i][j] = 0.0;
			Eps[i][j] = 0.0;
			D_eff[i][j] = 0.0;
		//	mu[i][j] = 1.0e7;
			Nt[i][j] = 0.0;
			Ind[i][j] = 0;

		}
	}
       
	rho_surf = rho[0][0];

	for (i = 0; i < Im; i++){
           	for (j = 1; j < J_max-1; j++){
           		if (j > Jm[i]) {
           			u[i][j] = 0.0;
           			v[i][j] = 0.0;
					mu[i][j] = 1.0;					
					p[i][j] = p0;
					rho[i][j] = StateEq1(p[i][j], Cg[i][j], rho0, c0, Gam_lq, p0, var_Eq_state);

              	}
            }
        }

	sprintf(out_name, "DataI/P");
	cut_file = fopen( out_name, "wt" );
	i_out = 0;
	j_out = 900;

	do
	{
		Tm = Tm + tau;
        printf("Time %5.6lf s \n", Tm);


        // ����� u

        for (i = 1; i < Im-1; i++){
        	for (j = 1; j < J_max-1; j++){
        		u_n[i][j] = u[i][j] + tau * ( 	-u[i][j] * (u[i+1][j] - u[i-1][j])/(2.0*hx)
        										-v[i][j] * (u[i][j+1] - u[i][j-1])/(2.0*hz) -
        										1.0/rho[i][j] * (p[i+1][j] - p[i][j])/hx +
        		 mu[i][j]/rho[i][j] * ( (u[i+1][j] -2.0 * u[i][j] +u[i-1][j])/(hx*hx) +
        				(u[i][j+1] -2.0 * u[i][j] +u[i][j-1])/(hz*hz))	+
						1 / rho[i][j] * ( (mu[i+1][j] - mu[i][j]) / hx * (u[i+1][j] - u[i-1][j]) / (2.0*hx) + 
						(mu[i][j + 1] - mu[i][j]) / hz + (u[i][j + 1] - u[i][j - 1]) / (2.0*hz)));
        	}
        }

        for (i = 1; i < Im-1; i++){
        	for (j = 1; j < J_max-1; j++){
        		u[i][j] = u_n[i][j];
        	}
        }
        // ��� ���������
       	for (j = 1; j < J_max-1; j++){
             u[0][j] = 0.0;
        }


        for (i = 1; i < Im-1; i++){
        	for (j = 1; j < J_max-1; j++){
				v_n[i][j] = v[i][j] + tau * (-10.0 - u[i][j] * (v[i + 1][j] - v[i - 1][j]) / (2.0*hx)
        										-v[i][j] * (v[i][j+1] - v[i][j-1])/(2.0*hz) -
        										1.0/rho[i][j] * (p[i][j+1] - p[i][j])/hz
        		+ mu[i][j]/rho[i][j] * ( (v[i+1][j] -2.0 * v[i][j] +v[i-1][j])/(hx*hx) +
        				(v[i][j+1] - 2.0 * v[i][j] + v[i][j-1])/(hz*hz))	+
						1 / rho[i][j] * ((mu[i + 1][j] - mu[i][j]) / hx * (v[i + 1][j] - v[i - 1][j]) / (2.0*hx) +
						(mu[i][j + 1] - mu[i][j]) / hz + (v[i][j + 1] - v[i][j - 1]) / (2.0*hz)));
        	}
        }

        for (i = 1; i < Im-1; i++){
        	for (j = 1; j < J_max-1; j++){
        		v[i][j] = v_n[i][j];
        	}
        }

        // ��� ���������
       	for (j = 1; j < J_max-1; j++){
             v[0][j] = v_n[1][j];
        }
		
		for (i = 0; i < Im; i++){
			v[i][0] = v[i][1];
		}

       	if (var_wall == 1) {
          for (j = 1; j < J_max-1; j++){
                 u_av = 0.0;
            for (i = 0; i < Im-1; i++){
            	 u_av =  u_av + v[i][j];
            }
            u_av = u_av / (Im-1);
			if (Cg[Im-2][j] < 0.4 ) {v[Im-1][j] = u_av * Cg[Im-2][j] / (1.0 - Cg[Im-2][j]);}
			else  {v[Im-1][j] = u_av * Cg[Im-2][j];};
  	      }
       	}

        // ����� rho

        for (i = 1; i < Im-1; i++){
          	for (j = 1; j < J_max-1; j++){
          		rho_n[i][j] = rho[i][j] + tau * (-u[i][j] * (rho[i+1][j] - rho[i-1][j])/(2.0*hx)
          										 -v[i][j] * (rho[i][j+1] - rho[i][j-1])/(2.0*hz) -
          					rho[i][j] * ((u[i][j] - u[i-1][j])/hx + (v[i][j] - v[i][j-1])/hz ));
				if (rho_n[i][j] < rho0 *0.25) { rho_n[i][j] = rho0 *0.25; };
          	}
          }

        // ��� ���������  � �������
       	for (j = 1; j < J_max-1; j++){
             rho_n[0][j] = rho_n[1][j];
             rho_n[Im-1][j] = rho_n[Im-2][j];
        }

        // ����� p

        for (i = 1; i < Im-1; i++){
        	for (j = 1; j < J_max-1; j++){
        		rho[i][j] = rho_n[i][j];
				p[i][j] = StateEq(rho[i][j], Cg[i][j], rho0, c0, Gam_lq, p0, var_Eq_state);
        	}
        }

		        // ��� ���������  � �������
       	for (j = 1; j < J_max-1; j++){
             p[0][j] = p[1][j];
			 rho[0][j] = StateEq1(p[0][j], Cg[0][j], rho0, c0, Gam_lq, p0, var_Eq_state);
             p[Im-1][j] = p[Im-2][j];
			 rho[Im - 1][j] = StateEq1(p[Im - 1][j], Cg[Im - 1][j], rho0, c0, Gam_lq, p0, var_Eq_state);
        }

		// Nucleation

		for (i = 0; i < Im; i++){
			for (j = 1; j <= Jm[i]; j++){
				switch(Ind[i][j])
				{ 
					// Nucleation
				case 0 : {		
					P_eq = Cp[i][j] * Cp[i][j] / (Kh*Kh);
					dP = P_eq - p[i][j];
					if (dP > 0.0) 
					{
						W_cr = 16.0*Pi*sigma*sigma*sigma / (3.0*dP*dP);
						kapp = 3.0;
						V_0 = 1.0e-6;
						N_0 = rho0 * Cp[i][j] / M_m;
						V_m = 0.018 / (1000 * Na);
						d_m = pow(6.0 / (Pi * N_0), 1.0 / 3.0);
						J_e = 2.0*N_0*N_0*V_m*De / d_m * sqrt(sigma / (kB*T));
						J = J_e*exp(-W_cr / (kB*T));
						Nt[i][j] = Nt[i][j] + V_0*J*tau;
						if (Nt[i][j] > 1.0) 
						{
							Nb[i][j] = 0.6* pow((kapp*kapp*kapp - 1.0) / (kapp*kapp*kapp*kapp*kapp) * J / De, 3.0 / 5.0);
							Rb[i][j] = 2.0*sigma / (dP);
							Pg[i][j] = p[i][j];
							Mg[i][j] = 4.0*Pi * Pg[i][j] * Rb[i][j] * Rb[i][j] * Rb[i][j] * MH2O / (3.0* Rg * T);
							Cp[i][j] = Cp0[i][j] - Nb[i][j] * Mg[i][j] / rho0;
							Cg[i][j] = Nb[i][j] * 4.0*Pi / 3.0 *Rb[i][j] * Rb[i][j] * Rb[i][j];
							//Cg[i][j] = Cg[i][j] / (1.0 + Cg[i][j]);
							E = E0*(1 - 12 * Cp[i][j]);
							mu[i][j] = mu0*exp(E / (kB*T));
						//	mu[i][j] = 1.0e7;
							rho[i][j] = StateEq1(p[i][j], Cg[i][j], rho0, c0, Gam_lq, p0, var_Eq_state);
							Ind[i][j] = 1;
						}
				}
				}
				break;
					// Grows bubbles 
				case 1: {
					dRb[i][j] = Rb[i][j] * (Pg[i][j] - p[i][j]) / (4.0* mu[i][j]);
					Rb[i][j] = Rb[i][j] + dRb[i][j] * tau;
					if (Rb[i][j] < Rb_min) { Rb[i][j] = Rb_min; };
					if (p[i][j] > p0) { Cp_temp = Kh*sqrt(p[i][j]); }
					else { Cp_temp = Kh*sqrt(p0); };
					dMg = 4.0*Pi* Rb[i][j] * rho0 * De * (Cp[i][j] - Cp_temp) *tau;
					if ((dMg > 0.0) && (dMg*Nb[i][j] > Cp[i][j] * rho0)) { dMg = Cp[i][j] * rho0 / Nb[i][j]; };
					//	if ((dMg < 0.0) && (dMg > Cp[i][j] * rho0)) { dMg = Cp[i][j] * rho0 / Nb[i][j]; };
					Mg[i][j] = Mg[i][j] + dMg;
					if (Mg[i][j] < 0.0) { Mg[i][j] = 4.0*Pi * p[i][j] * Rb[i][j] * Rb[i][j] * Rb[i][j] * MH2O / (3.0* Rg * T); };
					Pg[i][j] = 3.0* Mg[i][j] * Rg * T / (4.0*Pi *Rb[i][j] * Rb[i][j] * Rb[i][j] * MH2O);
					Cp[i][j] = Cp0[i][j] - Nb[i][j] * Mg[i][j] / rho0;
					if (Cp[i][j] < 0) { Cp[i][j] = Kh*sqrt(p0); };
					E = E0*(1 - 12 * Cp[i][j]);
					mu[i][j] = mu0*exp(E / (kB*T));			
				//	mu[i][j] = 1.0e7;
					Cg[i][j] = Nb[i][j] * 4.0*Pi / 3.0 *Rb[i][j] * Rb[i][j] * Rb[i][j];
					Cg[i][j] = Cg[i][j] / (1.0 + Cg[i][j]);
				//	if (Cg[i][j] > 0.75) { Cg[i][j] = 0.75; };
					p[i][j] = StateEq(rho[i][j], Cg[i][j], rho0, c0, Gam_lq, p0, var_Eq_state);
				}
						break;
				case 2: {
					if (p[i][j] > p0) { Eps[i][j] = rho0 / (MH2O * p[i][j] / (Rg * T))* (Cp[i][j] - Kh*sqrt(p[i][j])); }
					else { Eps[i][j] = rho0 / (MH2O * p0 / (Rg * T))* (Cp[i][j] - Kh*sqrt(p0)); };
					if (Eps[i][j] < 0.0) { Eps[i][j] = 0.0; };
					if (Eps[i][j] > 2.0) { D_eff[i][j] = De*12.0 / Pi * Eps[i][j] * Eps[i][j]; }
					else { D_eff[i][j] = De * 2.0 * Eps[i][j]; };
					Rb[i][j] = Rb[i][j] + tau*D_eff[i][j] / (2.0*Rb[i][j]);
					if (Rb[i][j] < Rb_min) { Rb[i][j] = Rb_min; };
					if (p[i][j] > p0) { Pg[i][j] = p[i][j]; }
					else {
						Pg[i][j] = p0;
					}
					Mg[i][j] = (MH2O * Pg[i][j] / (Rg * T)) * 4.0*Pi / 3.0 *Rb[i][j] * Rb[i][j] * Rb[i][j]; 
					Cp[i][j] = Cp0[i][j] - Nb[i][j] * Mg[i][j] / rho0;
					if (Cp[i][j] < 0) { Cp[i][j] = Kh*sqrt(p0); };
					Cg[i][j] = Nb[i][j] * 4.0*Pi / 3.0 *Rb[i][j] * Rb[i][j] * Rb[i][j];
					Cg[i][j] = Cg[i][j] / (1.0 + Cg[i][j]);
					p[i][j] = StateEq(rho[i][j], Cg[i][j], rho0, c0, Gam_lq, p0, var_Eq_state);
					E = E0*(1 - 12 * Cp[i][j]);
					mu[i][j] = mu0*exp(E / (kB*T));
				} 
						break;
				case 3: {
					ddRb[i][j] = (Pg[i][j] - p[i][j]) / (Rb[i][j] * rho[i][j]) -
						(4.0* mu[i][j] * dRb[i][j]) / (Rb[i][j] * rho[i][j] * Rb[i][j]) -
						3.0*dRb[i][j] * dRb[i][j] / (2.0*Rb[i][j]);
					dRb[i][j] = dRb[i][j] + ddRb[i][j] * tau;
					Rb[i][j] = Rb[i][j] + dRb[i][j] * tau;
					if (Rb[i][j] < Rb_min) { Rb[i][j] = Rb_min; };
					if (p[i][j] > p0) { Cp_temp = Kh*sqrt(p[i][j]); }
					else { Cp_temp = Kh*sqrt(p0); };
					dMg = 4.0*Pi* Rb[i][j] * rho0 * De * (Cp[i][j] - Cp_temp) *tau;
					if ((dMg > 0.0) && (dMg*Nb[i][j] > Cp[i][j] * rho0)) { dMg = Cp[i][j] * rho0 / Nb[i][j]; };
					//	if ((dMg < 0.0) && (dMg > Cp[i][j] * rho0)) { dMg = Cp[i][j] * rho0 / Nb[i][j]; };
					Mg[i][j] = Mg[i][j] + dMg;
					if (Mg[i][j] < 0.0) { Mg[i][j] = 4.0*Pi * p[i][j] * Rb[i][j] * Rb[i][j] * Rb[i][j] * MH2O / (3.0* Rg * T); };
					Pg[i][j] = 3.0* Mg[i][j] * Rg * T / (4.0*Pi *Rb[i][j] * Rb[i][j] * Rb[i][j] * MH2O);
					Cp[i][j] = Cp0[i][j] - Nb[i][j] * Mg[i][j] / rho0;
					if (Cp[i][j] < 0) { Cp[i][j] = Kh*sqrt(p0); };
					E = E0*(1 - 12 * Cp[i][j]);
					mu[i][j] = mu0*exp(E / (kB*T));
					Cg[i][j] = Nb[i][j] * 4.0*Pi / 3.0 *Rb[i][j] * Rb[i][j] * Rb[i][j];
					Cg[i][j] = Cg[i][j] / (1.0 + Cg[i][j]);
					//	if (Cg[i][j] > 0.75) { Cg[i][j] = 0.75; };
					p[i][j] = StateEq(rho[i][j], Cg[i][j], rho0, c0, Gam_lq, p0, var_Eq_state);
				} break;

				}
		
			}
		}


 /*     	for (i = 0; i < Im; i++){
			for (j = 1; j <= Jm[i]; j++){
				dRb[i][j] = Rb[i][j] * (Pg[i][j] - p[i][j]) / (4.0* mu[i][j]);
				Rb[i][j] = Rb[i][j] + dRb[i][j] * tau;
				if (Rb[i][j] < Rb_min) { Rb[i][j] = Rb_min; };
				if (p[i][j] > p0) {Cp_temp = Kh*sqrt(p[i][j]);}
				else {Cp_temp = Kh*sqrt(p0);};
				dMg = 4.0*Pi* Rb[i][j] * rho0 * De * (Cp[i][j] - Cp_temp) *tau;
				if ((dMg > 0.0) && (dMg*Nb[i][j] > Cp[i][j] * rho0)) { dMg = Cp[i][j] * rho0 / Nb[i][j]; };
			//	if ((dMg < 0.0) && (dMg > Cp[i][j] * rho0)) { dMg = Cp[i][j] * rho0 / Nb[i][j]; };
				Mg[i][j]= Mg[i][j] + dMg;
				if (Mg[i][j] < 0.0) { Mg[i][j] = 4.0*Pi * p[i][j] * Rb[i][j] * Rb[i][j] * Rb[i][j] * MH2O / (3.0* Rg * T); };
				Pg[i][j] = 3.0* Mg[i][j] * Rg * T / (4.0*Pi *Rb[i][j] * Rb[i][j] * Rb[i][j]* MH2O);
				Cp[i][j] = Cp0[i][j] - Nb[i][j]* Mg[i][j]/rho0;
				if (Cp[i][j] < 0) {Cp[i][j] = Kh*sqrt(p0);};
				E = E0*(1-12*Cp[i][j]);
				mu[i][j] =mu0*exp(E/(kB*T));
				Cg[i][j] = Nb[i][j]* 4.0*Pi/3.0 *Rb[i][j]*Rb[i][j]*Rb[i][j];
				Cg[i][j] = Cg[i][j] / (1.0 + Cg[i][j]);
				if (Cg[i][j] > 0.75) { Cg[i][j] = 0.75; };
				p[i][j] = StateEq(rho[i][j], Cg[i][j], rho0, c0, Gam_lq, p0);

			}
		}
		*/

  /*    //     	for (i = 0; i < Im; i++){
     	//		for (j = 1; j <= Jm[i]; j++){
       	{{ i=i_out; j = j_out;
     				ddRb[i][j] = (Pg[i][j] - p[i][j]) / (Rb[i][j] * rho[i][j]) -
     						(4.0* mu[i][j]*dRb[i][j])/(Rb[i][j] * rho[i][j]*Rb[i][j]) -
     								3.0*dRb[i][j]*dRb[i][j]/(2.0*Rb[i][j]);
     				dRb[i][j] = dRb[i][j] + ddRb[i][j] * tau;
     				Rb[i][j] = Rb[i][j] + dRb[i][j] * tau;
     				if (Rb[i][j] < Rb_min) { Rb[i][j] = Rb_min; };
     				Pg[i][j] = Mg[i][j] / (4.0*Pi / 3.0 *Rb[i][j] * Rb[i][j] * Rb[i][j]) * Rg * T / MH2O;
     				Cp[i][j] = Cp0[i][j] - Nb[i][j]* Mg[i][j]/rho0;
     				if (Cp[i][j] < 0) {Cp[i][j] = Kh*sqrt(p0);};
     				Cg[i][j] = Nb[i][j]* 4.0*Pi/3.0 *Rb[i][j]*Rb[i][j]*Rb[i][j];
     				Cg[i][j] = Cg[i][j] / (1.0 + Cg[i][j]);
     				p[i][j] = StateEq(rho[i][j], Cg[i][j], rho0, c0, Gam_lq, p0);

     			}
     		}
*/
 /*       for (i = 0; i < Im; i++){
        	for (j = 1; j <= Jm[i]; j++){
        		if (p[i][j] > p0) {Eps[i][j] = rho0/ (MH2O * p[i][j] / (Rg * T))* (Cp[i][j] - Kh*sqrt(p[i][j]));}
        		else {Eps[i][j] = rho0/ (MH2O * p0 / (Rg * T))* (Cp[i][j] - Kh*sqrt(p0));};
				if (Eps[i][j] < 0.0 ) {Eps[i][j] = 0.0;};
           		if (Eps[i][j] > 2.0 ) {D_eff[i][j] = De*12.0/Pi * Eps[i][j] * Eps[i][j];}
           		else {D_eff[i][j] = De * 2.0 * Eps[i][j];};
        		Rb[i][j] = Rb[i][j] + tau*D_eff[i][j] / (2.0*Rb[i][j]);
        		if (Rb[i][j] < Rb_min) {Rb[i][j] = Rb_min;};
				if (p[i][j] > p0) {Mg[i][j] = (MH2O * p[i][j] / (Rg * T)) * 4.0*Pi/3.0 *Rb[i][j]*Rb[i][j]*Rb[i][j];}
				else {Mg[i][j] = (MH2O * p0 / (Rg * T)) * 4.0*Pi/3.0 *Rb[i][j]*Rb[i][j]*Rb[i][j];}
        		Cp[i][j] = Cp0[i][j] - Nb[i][j]* Mg[i][j]/rho0;
				if (Cp[i][j] < 0) {Cp[i][j] = Kh*sqrt(p0);};
        		Cg[i][j] = Nb[i][j]* 4.0*Pi/3.0 *Rb[i][j]*Rb[i][j]*Rb[i][j];
        		Cg[i][j] = Cg[i][j] / (1.0 + Cg[i][j]);
        		p[i][j] = StateEq(rho[i][j], Cg[i][j], rho0, c0, Gam_lq, p0);
        		E = E0*(1-12*Cp[i][j]);
        		mu[i][j] =mu0*exp(E/(kB*T));
        	}
        }
*/
            // ��� ��������� ������������
		
        GrZ[Ig-1] = GrZ[Ig-1] + v[Im-1][Jm[Im-1]]*tau;
        Jm[Im-1] = int (GrZ[Ig-1] / hz);
		for (l = Ig-2; l >= 0; l--){
			i= int (GrX[l] /hx);
        //  GrX[l] = GrX[l] + u[i][Jm[i]]*tau;
          GrZ[l] = GrZ[l] + v[i][Jm[i]]*tau;
		if (GrZ[l] > Z_max) {GrZ[l] = Z_max;};
		  if (GrZ[l] < GrZ[l+1]) {GrZ[l] = GrZ[l+1];};
          Jm[i] = int (GrZ[l] / hz);
		}


        for (i = 0; i < Im; i++){
           	for (j = 1; j < J_max; j++){
           		if (j > Jm[i]) {
           			u[i][j] = u[i][Jm[i]];
           			v[i][j] = v[i][Jm[i]];
					Nb[i][j] = Nb[i][Jm[i]];
					Rb[i][j] = Rb[i][Jm[i]];
           			Cp[i][j] = Cp[i][Jm[i]];
           			Cg[i][j] = Cg[i][Jm[i]];
					Mg[i][j] = Mg[i][Jm[i]];
					Pg[i][j] = Pg[i][Jm[i]];
					mu[i][j] = mu[i][Jm[i]];
					Ind[i][j] = Ind[i][Jm[i]];
					
			/*		u[i][j] = u[i][1000];
           			v[i][j] = v[i][1000];
           			Rb[i][j] = Rb[i][1000];
           			Cp[i][j] = Cp[i][1000];
           			Cg[i][j] = Cg[i][1000];
					mu[i][j] = mu[i][1000]; */
           			p[i][j] = FreeSurf(Tm, p[i][j]);
					rho[i][j] = StateEq1(p[i][j], Cg[i][j], rho0, c0, Gam_lq, p0, var_Eq_state);
              	}
            }
        }





        //

	//	if (Tm > 0.035) {T_out = tau; }
        // ������ ������
     //   if ( (Tm > out_num * T_out) || (Tm > 57.0e-3)){
		if (Tm > out_num * T_out) {

        if (out_num < 100000) {sprintf(out_name, "Data/%d", out_num);};
        if (out_num < 10000) {sprintf(out_name, "Data/0%d", out_num);};
        if (out_num < 1000) {sprintf(out_name, "Data/00%d", out_num);};
        if (out_num < 100) {sprintf(out_name, "Data/000%d", out_num);};
        if (out_num < 10) {sprintf(out_name, "Data/0000%d", out_num);};
        out_file = fopen( out_name, "wt" );
    	for (i = 0; i < Im; i++){
    		for (j = 0; j <= Jm[i]; j++){
    			fprintf( out_file, "%10.8lf \t %10.8lf \t %10.8lf \t %10.8lf \t %10.8lf \t %10.8lf \t %10.8lf \t %10.8lf \t %10.8lf \t %10.8lf \t %10.8lf \t %10.8lf \t %10.8lf \n",
    					x[i], z[j], p[i][j]/p0, rho[i][j], Cg[i][j], mu[i][j], u[i][j], v[i][j], Cp[i][j], Rb[i][j]*1.0e6, dRb[i][j], Pg[i][j]/p0, Nb[i][j]);
    		}
    		fprintf( out_file, "\n");
    	}
    	fclose( out_file );

    	if (out_num < 100000) {sprintf(out_name, "DataG/%d", out_num);};
    	if (out_num < 10000) {sprintf(out_name, "DataG/0%d", out_num);};
    	if (out_num < 1000) {sprintf(out_name, "DataG/00%d", out_num);};
    	if (out_num < 100) {sprintf(out_name, "DataG/000%d", out_num);};
    	if (out_num < 10) {sprintf(out_name, "DataG/0000%d", out_num);};
    	out_file = fopen( out_name, "wt" );
    	for (l = 0; l < Ig; l++){
			i= int (GrX[l] /hx);
    		fprintf( out_file, "%10.8lf \t %10.8lf \t %d \n",GrX[l], GrZ[l], Jm[i]);
    		}
    	fclose( out_file );

    	if (out_num < 100000) {sprintf(out_name, "DataB/%d", out_num);};
    	if (out_num < 10000) {sprintf(out_name, "DataB/0%d", out_num);};
    	if (out_num < 1000) {sprintf(out_name, "DataB/00%d", out_num);};
    	if (out_num < 100) {sprintf(out_name, "DataB/000%d", out_num);};
    	if (out_num < 10) {sprintf(out_name, "DataB/0000%d", out_num);};
    	out_file = fopen( out_name, "wt" );
    	for (j = 0; j <= Jm[0]; j++){
			fprintf(out_file, "%10.8lf \t %10.8lf \t %10.8lf \t %10.8lf \t %10.8lf \t %10.8lf \n", 
				z[j], p[0][j] / p0, v[0][j], Nb[0][j], Cp[0][j], Cg[0][j]);
    		}
    	fclose( out_file );


    	out_num = out_num + 1;
      	

    	fprintf( cut_file, "%lf \t %lf \t %lf \t %lf \t %lf \t %lf \t %e \t %e \t %e \t %e \t %lf \t %e\n",
    	    					Tm*1.0e3, p[i_out][j_out]/p0, mu[i_out][j_out], Cp[i_out][j_out],
    	    					Eps[i_out][j_out],D_eff[i_out][j_out]/De, Rb[i_out][j_out]*1.0e6, Mg[i_out][j_out],
    	    					Cg[i_out][j_out], dRb[i_out][j_out], Pg[i_out][j_out]/p0, Nt[i_out][j_out]);
	  }							
	}  while (Tm < T_all);

	fclose( cut_file );
}
